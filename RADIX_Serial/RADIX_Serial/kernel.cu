﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <algorithm>  // Para std::max_element

using namespace std;

int getDigit(int num, int digitPosition) {
    return (num / static_cast<int>(pow(10, digitPosition))) % 10;
}

void countingSort(vector<int>& arr, int digitPosition) {
    const int base = 10;
    vector<int> count(base, 0);
    vector<int> output(arr.size());

    for (int i = 0; i < arr.size(); i++) {
        int digit = getDigit(arr[i], digitPosition);
        count[digit]++;
    }

    for (int i = 1; i < base; i++) {
        count[i] += count[i - 1];
    }

    for (int i = arr.size() - 1; i >= 0; i--) {
        int digit = getDigit(arr[i], digitPosition);
        output[count[digit] - 1] = arr[i];
        count[digit]--;
    }

    for (int i = 0; i < arr.size(); i++) {
        arr[i] = output[i];
    }
}

void radixSort(vector<int>& arr) {
    int maxElement = *max_element(arr.begin(), arr.end());
    int maxDigits = log10(maxElement) + 1;

    for (int i = 0; i < maxDigits; i++) {
        countingSort(arr, i);
    }
}

int main() {
    // Inicializando a semente para números aleatórios com o tempo atual
    srand(static_cast<unsigned int>(time(0)));

    vector<int> sizes = { 100, 1000, 10000, 100000, 1000000, 10000000 };

    for (int size : sizes) {
        vector<int> arr(size);
        for (int i = 0; i < size; i++) {
            arr[i] = rand() % 1000000;  // Números aleatórios entre 0 e 999999
        }

        clock_t start = clock();
        radixSort(arr);
        clock_t end = clock();

        double duration = double(end - start) / CLOCKS_PER_SEC;
        cout << "Array Size: " << size << ", Execution Time: " << duration << " s" << endl;
    }

    return 0;
}