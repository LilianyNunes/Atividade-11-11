#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <algorithm>
#include <hip/hip_runtime.h>

using namespace std;

__device__ int getDigit(int num, int digitPosition) {
    return (num / static_cast<int>(pow(10, digitPosition))) % 10;
}

__global__ void countingSortKernel(int* arr, int* output, int* count, int size, int digitPosition) {
    const int base = 10;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Contagem dos dígitos
    if (idx < size) {
        int digit = getDigit(arr[idx], digitPosition);
        atomicAdd(&count[digit], 1); // Count each digit per thread
    }
    __syncthreads();

    // Realiza a contagem acumulada de forma segura
    if (idx == 0) {
        for (int i = 1; i < base; i++) {
            count[i] += count[i - 1]; // Accumulate counts
        }
    }
    __syncthreads();

    // Coloca os elementos na posição correta com base no dígito atual
    if (idx < size) {
        int digit = getDigit(arr[idx], digitPosition);
        int position = atomicAdd(&count[digit], -1) - 1; // Find position for the element
        output[position] = arr[idx]; // Place the element in the correct position
    }
}

void radixSort(vector<int>& arr) {
    int maxElement = *max_element(arr.begin(), arr.end());
    int maxDigits = log10(maxElement) + 1;

    int size = arr.size();
    int* d_arr, * d_output, * d_count;

    // Allocate memory on the device
    hipMalloc(&d_arr, size * sizeof(int));
    hipMalloc(&d_output, size * sizeof(int));
    hipMalloc(&d_count, 10 * sizeof(int));

    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_count, 0, 10 * sizeof(int));

    int blockSize = 256; // Tamanho do bloco de threads
    int gridSize = (size + blockSize - 1) / blockSize;

    // Para cada dígito, realiza o countingSort paralelizado
    for (int i = 0; i < maxDigits; i++) {
        hipMemset(d_count, 0, 10 * sizeof(int)); // Limpa o contador de dígitos antes de cada iteração
        countingSortKernel << <gridSize, blockSize >> > (d_arr, d_output, d_count, size, i);
        hipMemcpy(arr.data(), d_output, size * sizeof(int), hipMemcpyDeviceToHost); // Copia de volta os dados ordenados
    }

    hipFree(d_arr);
    hipFree(d_output);
    hipFree(d_count);
}

int main() {
    srand(static_cast<unsigned int>(time(0)));

    vector<int> sizes = { 100, 1000, 10000, 100000, 1000000, 10000000 };

    for (int size : sizes) {
        vector<int> arr(size);
        for (int i = 0; i < size; i++) {
            arr[i] = rand() % 1000000;  // Números aleatórios entre 0 e 999999
        }

        clock_t start = clock();
        radixSort(arr);
        clock_t end = clock();

        double duration = double(end - start) / CLOCKS_PER_SEC;
        cout << "Array Size: " << size << ", Execution Time: " << duration << " s" << endl;
    }

    return 0;
}
